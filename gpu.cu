#ifndef GPU_C
#define GPU_C

#include <stdio.h>

// CUDA Kernel function to add two numbers
__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int calculate()
{
    int a, b, c; // Host copies of a, b, c
    int *d_c;    // Device copy of c
    int size = sizeof(int);

    // Setup input values
    a = 2;
    b = 7;

    // Allocate space for device copy of c
    hipMalloc((void **)&d_c, size);

    // Launch add() kernel on GPU with one thread
    add<<<1, 1>>>(a, b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_c);

    printf("Result: %d + %d = %d\n", a, b, c);
    return 0;
}

#endif